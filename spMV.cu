#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

int main() {
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;

    // Initialize cuSPARSE library
    hipsparseCreate(&handle);

    // Create stream and assign to handle
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipsparseSetStream(handle, stream);

    // Define a simple CSR matrix
    // | 1 0 0 |
    // | 0 2 0 |
    // | 4 0 3 |
    int h_csrRowPtr[] = {0, 1, 2, 4};
    int h_csrColInd[] = {0, 1, 0, 2};
    float h_csrVal[] = {1.0f, 2.0f, 4.0f, 3.0f};
    float h_x[] = {1.0f, 2.0f, 3.0f};
    float h_y[3] = {0};

    // Device arrays
    int *d_csrRowPtr, *d_csrColInd;
    float *d_csrVal, *d_x, *d_y;

    // Allocate device memory
    hipMalloc(&d_csrRowPtr, sizeof(h_csrRowPtr));
    hipMalloc(&d_csrColInd, sizeof(h_csrColInd));
    hipMalloc(&d_csrVal, sizeof(h_csrVal));
    hipMalloc(&d_x, sizeof(h_x));
    hipMalloc(&d_y, sizeof(h_y));

    // Copy data to device
    hipMemcpy(d_csrRowPtr, h_csrRowPtr, sizeof(h_csrRowPtr), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, h_csrColInd, sizeof(h_csrColInd), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, h_csrVal, sizeof(h_csrVal), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice);

    // Create matrix and vector descriptors
    hipsparseCreateCsr(&matA, 3, 3, 4, d_csrRowPtr, d_csrColInd, d_csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, 3, d_x, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, 3, d_y, HIP_R_32F);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Determine buffer size required for the hipsparseSpMV operation
    size_t bufferSize = 0;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);

    // Allocate the buffer
    void* dBuffer = NULL;
    hipMalloc(&dBuffer, bufferSize);

    // Perform matrix-vector multiplication: y = A*x
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // Copy result back to host
    hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < 3; i++) {
        std::cout << "y[" << i << "] = " << h_y[i] << std::endl;
    }

    // Cleanup
    hipFree(d_csrRowPtr);
    hipFree(d_csrColInd);
    hipFree(d_csrVal);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(dBuffer);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    return 0;
}
